#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define BLOCK_WIDTH 16 
#define TILE_WIDTH  BLOCK_WIDTH

extern "C" void gpu_mat_mul(float* h_M, float* h_N, float* h_P, int width);

__global__
void gpu_mat_mul_kernel(float* M, float* N, float* P, int width){

  __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x; 
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Identify the row and column of the P element to work on
  // Each thread works on an element of P
  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;

  float sum = 0;
  int phase_num = width/TILE_WIDTH;

  // Each thread loads 'Row'th row of M and 'Col'th column of N
  for (int ph = 0; ph < phase_num; ph++) {    

    // Collaboratively load data into shared memory
    Mds[ty][tx] = M[Row * width + ph * TILE_WIDTH + tx];   
    Nds[ty][tx] = N[(ph * TILE_WIDTH + ty) * width + Col];

    __syncthreads();
    for (int k = 0; k < TILE_WIDTH; k++) { 
      sum += Mds[ty][k] * Nds[k][tx];
    }
    __syncthreads();
  }

  P[Row * width + Col] = sum;
}

void gpu_mat_mul(float* h_M, float* h_N, float* h_P, int width) {
  float *d_M, *d_N, *d_P;

  size_t size_of_float = sizeof(float);
  size_t size_M = width * width * size_of_float;
  size_t size_N = width * width * size_of_float;
  size_t size_P = width * width * size_of_float;

  hipMalloc((void**)&d_M, size_M);
  hipMalloc((void**)&d_N, size_N);
  hipMalloc((void**)&d_P, size_P);
    
  hipMemcpy(d_M, h_M, size_M, hipMemcpyHostToDevice);
  hipMemcpy(d_N, h_N, size_N, hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  float elapsed_time = 0.0;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  dim3 grid_dim(width/BLOCK_WIDTH, width/BLOCK_WIDTH, 1);
  dim3 block_dim(BLOCK_WIDTH, BLOCK_WIDTH, 1);
  gpu_mat_mul_kernel<<<grid_dim, block_dim>>>(d_M, d_N, d_P, width);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(h_P, d_P, size_P, hipMemcpyDeviceToHost);
    
  // Free device memory for M, N, P
  hipFree(d_M);
  hipFree(d_N);
  hipFree(d_P);

  hipEventElapsedTime(&elapsed_time, start, stop);
    
  printf("  grid  dim:  %d, %d, %d.\n", grid_dim.x, grid_dim.y, grid_dim.z);
  printf("  block dim: %d, %d, %d.\n", block_dim.x, block_dim.y, block_dim.z);
  printf("  kernel time: %.5f sec\n", elapsed_time / 1000);

  hipEventDestroy(start);
  hipEventDestroy(stop);
}


